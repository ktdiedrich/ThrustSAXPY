#include <gtest/gtest.h>
#include <hipDNN.h>

TEST(CuDNNTest, CanCreateHandle) {
    hipdnnHandle_t handle;
    hipdnnStatus_t status = hipdnnCreate(&handle);
    EXPECT_EQ(status, HIPDNN_STATUS_SUCCESS) << "cuDNN handle creation failed: " << hipdnnGetErrorString(status);
    if (status == HIPDNN_STATUS_SUCCESS) {
        hipdnnDestroy(handle);
    }
}

TEST(CuDNNTest, VersionIsNonZero) {
    EXPECT_GT(CUDNN_VERSION, 0);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}