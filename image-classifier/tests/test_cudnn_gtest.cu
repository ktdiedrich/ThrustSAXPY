#include <gtest/gtest.h>
#include <hipDNN.h>
#include "minimal.h"
#include "graph.h"


TEST(CuDNNTest, CanCreateHandle) {
    hipdnnHandle_t handle;
    hipdnnStatus_t status = hipdnnCreate(&handle);
    EXPECT_EQ(status, HIPDNN_STATUS_SUCCESS) << "cuDNN handle creation failed: " << hipdnnGetErrorString(status);
    if (status == HIPDNN_STATUS_SUCCESS) {
        hipdnnDestroy(handle);
    }
}

TEST(CuDNNTest, VersionIsNonZero) {
    EXPECT_GT(CUDNN_VERSION, 0);
}

TEST(CuDNNTest, SigmoidActivationRuns) {
    // This will run your minimal sigmoid activation example.
    // If it doesn't throw or crash, the test passes.
    sigmoid_activate_tensor();
    SUCCEED();
}


TEST(CuDNNTest, GraphCreation) {
    // Test if the graph can be created without errors.
    create_graph();
    SUCCEED();
}


int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}